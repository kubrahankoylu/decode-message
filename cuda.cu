#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <time.h>


__global__
void findResult(char *inputFile, char* outputFile)
{
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	int n = 0;

	for (int i = (idx*100); i <= ((idx*100) + 99); i++) {
		
		if (*(inputFile + i) == ',') {
			*(outputFile + (4 * idx + n)) = *(inputFile + (i+1));
			n++;
		}
	}

}

int main(int argc, char** argv)
{
	clock_t time;
	FILE* f = fopen(argv[1], "r");
	char* inputFile_h = (char*)malloc(1550000 * sizeof(char));
	int i, j;
	i = 0;

	while ((j = fgetc(f)) != EOF) {
		*(inputFile_h + i) = (char)j;
		i++;
	}
	fclose(f);
	*(inputFile_h + i) = '\0';

	printf("serial(s) or parallel(p): \n");
	char choice;
	scanf("%c", &choice);

	char* outputFile_h = (char*)malloc(1000000 * sizeof(char));
	
	if (choice == 's') {
		int k = 0;
		i = 0;
		char ch;

		time = clock();

		while (*(inputFile_h + i) != '\0') {
			ch = (*(inputFile_h + i));
			if (ch == ',') {
				*(outputFile_h + k) = *(inputFile_h + (i + 1));
				k++;
			}
			i++;
		}
		*(outputFile_h + k) = '\0';
	}

	else if (choice == 'p') {
		char* inputFile_d = '\0';
		char* outputFile_d = '\0';

		size_t i_size = (strlen(inputFile_h)) * sizeof(char);
		size_t o_size = 1000000 * sizeof(char);

		time = clock();
		hipMalloc((void**)&inputFile_d, i_size);
		hipMalloc((void**)&outputFile_d, o_size);		

		hipMemcpy(inputFile_d, inputFile_h, i_size, hipMemcpyHostToDevice);

		findResult << < 15, 1024 >> > (inputFile_d, outputFile_d);
		
		hipMemcpy(outputFile_h, outputFile_d, o_size, hipMemcpyDeviceToHost);

	}
	hipDeviceSynchronize();
	time = clock() - time;
	f = fopen("output.txt", "w");
	
	for (int i = 0; i < 61441; i++) {
		if (*(outputFile_h + i) != NULL) {
			fprintf(f, "%c", *(outputFile_h + i));
		}
		
	}
	fclose(f);
	printf("Time: %f second\n", ((float)time / CLOCKS_PER_SEC));

	system("PAUSE");
}